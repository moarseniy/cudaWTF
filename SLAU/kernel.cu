#include "hip/hip_runtime.h"
//BETA TEST SOLVE SLAE SPARSE MATRIX
#include <stdio.h>
#include <stdlib.h>
//#include <iostream>
#include <assert.h>
#include "hip/device_functions.h"
#include ""
#include <hip/hip_runtime.h>
#include <hipsparse.h>


hipsparseHandle_t handle;


hipsparseMatDescr_t descrA = 0;
hipsparseMatDescr_t descr_L = 0;
hipsparseMatDescr_t descr_U = 0;

csrilu02Info_t info_A = 0;
csrsv2Info_t info_L = 0;
csrsv2Info_t info_U = 0;

void* pBuffer = 0;


void setUpLU(hipsparseMatDescr_t& descrLU, hipsparseMatrixType_t matrixType, hipsparseIndexBase_t indexBase, hipsparseFillMode_t fillMode, hipsparseDiagType_t diagType) {
    hipsparseCreateMatDescr(&descrLU);
    hipsparseSetMatType(descrLU, matrixType);
    hipsparseSetMatIndexBase(descrLU, indexBase);
    hipsparseSetMatFillMode(descrLU, fillMode);
    hipsparseSetMatDiagType(descrLU, diagType);
}
 
void memoryLU(csrilu02Info_t& info_A, csrsv2Info_t& info_L, csrsv2Info_t& info_U, hipsparseHandle_t handle, const int N, const int nnz, hipsparseMatDescr_t descrA, hipsparseMatDescr_t descr_L,
    hipsparseMatDescr_t descr_U, double* d_A, int* d_A_RowIndices, int* d_A_ColIndices, hipsparseOperation_t matrixOperation, void** pBuffer) {

    hipsparseCreateCsrilu02Info(&info_A);
    hipsparseCreateCsrsv2Info(&info_L);
    hipsparseCreateCsrsv2Info(&info_U);

    int pBufferSize_M, pBufferSize_L, pBufferSize_U;
    hipsparseDcsrilu02_bufferSize(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, &pBufferSize_M);
    hipsparseDcsrsv2_bufferSize(handle, matrixOperation, N, nnz, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, &pBufferSize_L);
    hipsparseDcsrsv2_bufferSize(handle, matrixOperation, N, nnz, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, &pBufferSize_U);

    int pBufferSize = max(pBufferSize_M, max(pBufferSize_L, pBufferSize_U));
    hipMalloc((void**)pBuffer, pBufferSize);
    printf("razmer matric= %d %d %d ", pBufferSize_M, pBufferSize_L, pBufferSize_U);
    
}


void analysisLU(csrilu02Info_t& info_A, csrsv2Info_t& info_L, csrsv2Info_t& info_U, hipsparseHandle_t handle, const int N, const int nnz, hipsparseMatDescr_t descrA, hipsparseMatDescr_t descr_L,
    hipsparseMatDescr_t descr_U, double* d_A, int* d_A_RowIndices, int* d_A_ColIndices, hipsparseOperation_t matrixOperation, hipsparseSolvePolicy_t solvePolicy1, hipsparseSolvePolicy_t solvePolicy2, void* pBuffer) {

    int structural_zero;

    hipsparseDcsrilu02_analysis(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, solvePolicy1, pBuffer);
    hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(handle, info_A, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status) 
    { 
        printf("A(%d,%d) net\n", structural_zero, structural_zero);
    }

    hipsparseDcsrsv2_analysis(handle, matrixOperation, N, nnz, descr_L, d_A, d_A_RowIndices, d_A_ColIndices, info_L, solvePolicy1, pBuffer);
    
    hipsparseDcsrsv2_analysis(handle, matrixOperation, N, nnz, descr_U, d_A, d_A_RowIndices, d_A_ColIndices, info_U, solvePolicy2, pBuffer);

}

void computeSparseLU(csrilu02Info_t& info_A, hipsparseHandle_t handle, const int N, const int nnz, hipsparseMatDescr_t descrA, double* d_A, int* d_A_RowIndices,
    int* d_A_ColIndices, hipsparseSolvePolicy_t solutionPolicy, void* pBuffer) {

    int numerical_zero;

    hipsparseDcsrilu02(handle, N, nnz, descrA, d_A, d_A_RowIndices, d_A_ColIndices, info_A, solutionPolicy, pBuffer);
    hipsparseStatus_t status = hipsparseXcsrilu02_zeroPivot(handle, info_A, &numerical_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == status) 
    { 
        printf("U(%d,%d) is zero\n", numerical_zero, numerical_zero); 
    }

}


int main(void)
{
    
    hipsparseCreate(&handle);

    const int Nrows = 4;    //kolvo strok                  
    const int Ncols = 4;    //kolvo stolbcov                      
    const int N = Nrows;

    
    FILE* right = fopen("right.txt", "r");
    double* h_x = (double*)malloc(Nrows * sizeof(*h_x));
    for(int i = 0; i<Nrows; i++)
    {
        fscanf(right, "%lf", &h_x[i]);
        //printf("%lf ", h_x[i]);
        
    }
    fclose(right);

    double* d_x;
    hipMalloc(&d_x, Nrows * sizeof(*d_x));
    hipMemcpy(d_x, h_x, Nrows * sizeof(*h_x), hipMemcpyHostToDevice);

    

    hipsparseCreateMatDescr(&descrA);
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE);
    

    FILE* data = fopen("data.txt", "r");
    FILE* ind = fopen("ind.txt", "r");
    FILE* ptr = fopen("ptr.txt", "r");
    
    double* h_A = (double*)malloc(Nrows * Ncols * sizeof(*h_A));
    
    int nnz=0, k = 0;
    while(fscanf(data,"%lf",&h_A[k])==1)
    {
        //printf("%lf ",h_A[k]);
        nnz++;
        k++;
            
    }
    printf("non zero = %d \n",nnz);

    double* d_A;       
    hipMalloc(&d_A, nnz * sizeof(*d_A));
    hipMemcpy(d_A, h_A, nnz * sizeof(*h_A), hipMemcpyHostToDevice);

    
    
    int* d_ptr;    hipMalloc(&d_ptr, (Nrows + 1) * sizeof(*d_ptr));
    int* d_ind;    hipMalloc(&d_ind, nnz * sizeof(*d_ind));

    int* h_ptr = (int*)malloc((Nrows + 1) * sizeof(*h_ptr));
    int* h_ind = (int*)malloc(nnz * sizeof(*h_ind));
    k = 0;
    
    while (fscanf(ind, "%d", &h_ind[k]) == 1)
    {
        //printf("%lf ", h_ind[k]);
        k++;

    }
    k = 0;
    while (fscanf(ptr, "%d", &h_ptr[k]) == 1)
    {
        //printf("%lf ", h_ptr[k]);
        k++;

    }
    hipMemcpy(d_ptr, h_ptr, (Nrows + 1) * sizeof(*h_ptr), hipMemcpyHostToDevice);
    hipMemcpy(d_ind, h_ind, nnz * sizeof(*h_ind), hipMemcpyHostToDevice);
    fclose(data);fclose(ptr);fclose(ind);

    

    printf("\nCSR MATRIX:\n\n");
    for (int i = 0; i < nnz; ++i) 
        printf("A[%i] = %.0f ", i, h_A[i]); 
    printf("\n");

    printf("\n");
    for (int i = 0; i < (Nrows + 1); ++i) 
        printf("h_ptr[%i] = %i \n", i, h_ptr[i]); 
    printf("\n");

    for (int i = 0; i < nnz; ++i) 
        printf("h_ind[%i] = %i \n", i, h_ind[i]);
    
   
    
    setUpLU(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ONE, HIPSPARSE_FILL_MODE_LOWER, HIPSPARSE_DIAG_TYPE_UNIT);
    setUpLU(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL, HIPSPARSE_INDEX_BASE_ONE, HIPSPARSE_FILL_MODE_UPPER, HIPSPARSE_DIAG_TYPE_NON_UNIT);   
    memoryLU(info_A, info_L, info_U, handle, N, nnz, descrA, descr_L, descr_U, d_A, d_ptr, d_ind, HIPSPARSE_OPERATION_NON_TRANSPOSE, &pBuffer);
    

    analysisLU(info_A, info_L, info_U, handle, N, nnz, descrA, descr_L, descr_U, d_A, d_ptr, d_ind, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_SOLVE_POLICY_NO_LEVEL,HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer);
    
    /* PARASHA
    double* d_A1;
    double* h_A1 = (double*)malloc(nnz * sizeof(double));
    hipMalloc(&d_A1, nnz * sizeof(*d_A1));
    hipMemcpy(h_A1, d_A, nnz * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(d_A1, h_A1, nnz * sizeof(double), hipMemcpyHostToDevice);
    for (int i = 0; i < nnz; i++)
        printf("%lf ", h_A1[i]);*/


    //A = L * U 
    computeSparseLU(info_A, handle, N, nnz, descrA, d_A, d_ptr, d_ind, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);//TUT OSHIBKA GDE TO!!!!!!!!!!!!!!!!!!!!!!!
    

    double* d_ztest = (double*)malloc(nnz * sizeof(*d_ztest));
    hipMemcpy(d_ztest, d_A, nnz * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < nnz; i++)
        printf("%lf ", d_ztest[i]);


    //L * z = x 
    const double alpha = 1.;
    
    double* d_z; 
    hipMalloc(&d_z, N * sizeof(*d_z));
   
    hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, &alpha, descr_L, d_A, d_ptr, d_ind, info_L, d_x, d_z, HIPSPARSE_SOLVE_POLICY_NO_LEVEL, pBuffer);
    
    

    printf("\n");

    //U * y = z 
    double* d_y;        
    hipMalloc(&d_y, Ncols * sizeof(*d_y));

    hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, nnz, &alpha, descr_U, d_A, d_ptr, d_ind, info_U, d_z, d_y, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, pBuffer);


    //resultat
    double* h_y = (double*)malloc(Ncols * sizeof(*h_y));
    hipMemcpy(h_y, d_y, N * sizeof(double), hipMemcpyDeviceToHost);

    printf("\n\nRESULTAT\n");

    for (int k = 0; k < N; k++) 
        printf("x[%i] = %f\n", k, h_y[k]);

    hipFree(pBuffer);
    hipsparseDestroyMatDescr(descrA);
    hipsparseDestroyMatDescr(descr_L);
    hipsparseDestroyMatDescr(descr_U);
    hipsparseDestroyCsrilu02Info(info_A);
    hipsparseDestroyCsrsv2Info(info_L);
    hipsparseDestroyCsrsv2Info(info_U);
    hipsparseDestroy(handle);
}