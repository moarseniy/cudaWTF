
#include "hip/hip_runtime.h"

#include <ctime>
#include <stdio.h>
#include <ctime>

#define N 50000

void add_cpu(int* a, int* b, int* c)
{
	int tid = 0;
	while (tid < N)
	{
		c[tid] = a[tid] + b[tid];
		tid += 1;
	}
}

__global__ void add_gpu(int* a, int* b, int* c)
{
	int tid = blockIdx.x;
	if (tid<N)
		c[tid] = a[tid] + b[tid];
}
int main()
{
	hipEvent_t start, stop;
	float gpuTime = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//1 part
	int a[N];
	int b[N];
	int c[N];
	
	for (int i = 0; i < N; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}

	hipEventRecord(start, 0);
	add_cpu(a, b, c);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&gpuTime, start, stop);
	printf("GPU time = %.4f \n", gpuTime);
	gpuTime = 0.0;
	for (int i = 0; i < N; i++)
	{
		//printf("%d + %d =%d\n", a[i], b[i], c[i]);
	}
	


	//2 part
	int* dev_a;
	int* dev_b;
	int* dev_c;
	
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_c, N * sizeof(int));

	for (int i = 0; i < N; i++)
	{
		a[i] = -i;
		b[i] = i * i;
	}

	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	hipEventRecord(start, 0);

	add_gpu <<< N, 1 >>> (dev_a, dev_b, dev_c);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&gpuTime, start, stop);
	printf("GPU time = %.4f \n", gpuTime);

	hipMemcpy(dev_c, c, N * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++)
	{
		//printf("%d + %d =%d\n", a[i], b[i], c[i]);
	}

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
