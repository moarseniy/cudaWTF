#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include <iostream>
#include <fstream>

using namespace std;


hipEvent_t start, stop;


__global__ void func(char* strDev, char* referenceDev, int* dlinaDev, float* scoringDev, int* startDev, int ref_len, int reads_len, int n, int m, const int maximum_read)
{

	int stolb = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ char slice_reference[2561];//84//2136

	//int xstr = stolb % reads_len, ystr = stolb / reads_len;
	//int istr = ystr * reads_len + xstr;

	int sum = 0; int start = startDev[stolb];
	int total_slides = ref_len - dlinaDev[stolb] + 1;
	for (int stroka = 0; stroka < total_slides; stroka++) {
		for (int k = 0; k < dlinaDev[stolb]; k++) {
			slice_reference[k] = referenceDev[stroka + k];
		}
		for (int k = 0; k < dlinaDev[stolb]; k++) {
			sum += (int)(slice_reference[k] == strDev[start + k]);
		}
		scoringDev[stroka + stolb * n] = (float)sum / dlinaDev[stolb];
		sum = 0; //start = 0;
	}

}

/*
__global__ void func(char* strDev, char* referenceDev, int* dlinaDev, float* scoringDev, int ref_len)
{
	extern __shared__ char slice_reference[];
	int index = threadIdx.x;

	int xstr = index % dlinaDev[index], ystr = index / dlinaDev[index];
	int istr = ystr * dlinaDev[index] + xstr;

	int sum = 0; int start = 0;
	int total_slides = ref_len - dlinaDev[index] + 1;
	for (int i = 0; i < total_slides; i++) {
		for (int k = 0; k < index; k++) {
			start += dlinaDev[k];
		}
		for (int k = 0; k < dlinaDev[index]; k++) {
			slice_reference[k] = referenceDev[i + k];
		}
		for (int k = 0; k < dlinaDev[index]; k++) {
			sum +=(int)(slice_reference[k] == strDev[start + k]);
		}
		scoringDev[istr] = sum / dlinaDev[index];
		sum = 0;
	}

}*/


int minimum(int* dlina, int reads_count) {
	int min = dlina[0];
	for (int i = 1; i < reads_count; i++) {
		if (dlina[i] < min) {
			min = dlina[i];
		}
	}
	return min;
}

int maximum(int* dlina, int reads_count) {
	int max = dlina[0];
	for (int i = 1; i < reads_count; i++) {
		if (dlina[i] > max) {
			max = dlina[i];
		}
	}
	return max;
}


int main(void)
{
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float gpuTime = 0.0;

	ifstream dna, reads, lengths;
	ofstream out;
	out.open("output.txt");
	lengths.open("lengths.txt");
	dna.open("dna.txt");
	reads.open("reads.txt");

	int ref_len = 48502;//48502;
	int reads_len = 302094;//2056551;//302094;//2056551;//302094;//2056551;
	int reads_count = 900;//6000;

	int* dlina = (int*)malloc(reads_count * sizeof(int));
	for (int i = 0; i < reads_count; i++) {
		lengths >> dlina[i];
		//cout << dlina[i] << " ";
	}

	/*
	int number_elements = 0;
	for (int i = 0; i < reads_count; i++) {
		number_elements += dlina[i];
	}
	cout << "summa=" << number_elements << endl;
	*/

	char* reference = (char*)malloc(ref_len * sizeof(char));
	for (int i = 0; i < ref_len; i++) {
		dna >> reference[i];
	}
	//for (int i = 0; i < ref_len; i++) {
	//	cout << reference[i];
	//}

	/*
	char** str = (char**)malloc(6000 * sizeof(char*));
	for (int i = 0; i < 6000; i++) {
		str[i] = (char*)malloc(dlina[i] * sizeof(char));
	}

	for (int i = 0; i < 6000; i++) {
		for (int j = 0; j < dlina[i]; j++) {
			reads >> str[i][j];
		}
	}
	*/
	char* str = new char[reads_len];
	for (int i = 0; i < reads_len; i++) {
		reads >> str[i];
	}
	/*
	for (int j = 0; j < reads_len; j++) {
		cout << str[j];
	}*/

	int n_scoring = ref_len - minimum(dlina, reads_count) + 1;//48459;
	int m_scoring = reads_count;

	float* scoring_matrix = new float[n_scoring * m_scoring];
	for (int i = 0; i < n_scoring * m_scoring; i++) {
		scoring_matrix[i] = 0;
	}

	int maximum_read = maximum(dlina, reads_count);
	cout << maximum_read << endl;

	//char* slice_reference = new char[maximum_read];
	//cout << n_scoring * m_scoring;
	//cout << ref_len - minimum_read(dlina, reads_count) + 1 << endl;
	//cout << "min=" << minimum_read(dlina, reads_count) << endl;

	int sum = 0;
	int* start_values = (int*)malloc(reads_count * sizeof(int));
	for (int i = 0; i < reads_count; i++) {
		sum += dlina[i];
		start_values[i] = sum;
	}

	hipSetDevice(0);

	char* strDev;
	char* referenceDev;
	int* dlinaDev;
	float* scoringDev;
	int* startDev;

	hipMalloc((void**)&strDev, reads_len * sizeof(char));
	hipMalloc((void**)&referenceDev, ref_len * sizeof(char));
	hipMalloc((void**)&dlinaDev, reads_count * sizeof(int));
	hipMalloc((void**)&scoringDev, n_scoring * m_scoring * sizeof(float));
	hipMalloc((void**)&startDev, reads_count * sizeof(int));

	hipMemcpy(strDev, str, reads_len * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(referenceDev, reference, ref_len * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(dlinaDev, dlina, reads_count * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(startDev, start_values, reads_count * sizeof(int), hipMemcpyHostToDevice);


	hipEventRecord(start, 0);

	func << <3, 300 >> > (strDev, referenceDev, dlinaDev, scoringDev, startDev, ref_len, reads_len, n_scoring, m_scoring, maximum_read);





	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);
	printf("GPU time = %.4f \n", gpuTime);


	hipMemcpy((void*)scoring_matrix, scoringDev, n_scoring * m_scoring * sizeof(float), hipMemcpyDeviceToHost);


	
	for (int i = 0; i < n_scoring; i++) {
		for (int j = 0; j < m_scoring; j++) {
			out << scoring_matrix[i + j * n_scoring] << " ";
		}
		out << endl;
	}
	/*
	for (int i = 0; i < n_scoring; i++) {
		for (int j = 0; j < m_scoring; j++) {
			cout << scoring_matrix[i + j * n_scoring] << " ";
		}
		cout << endl;
	}*/

	cout << scoring_matrix[0] << endl;
	cout << scoring_matrix[1] << endl;
	cout << scoring_matrix[2] << endl;

	delete[] str;
	//delete[] reference;
	//delete[] dlina;
	delete[] scoring_matrix;

	hipFree(strDev);
	hipFree(referenceDev);
	hipFree(dlinaDev);
	hipFree(scoringDev);


	return 0;
}
